#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <conio.h>
#include <time.h>
#include <Windows.h>

/* ������� ��� ������� */
typedef struct
{
	int width;
	int height;
	double* elements;
} Matrix;


/*��������� �����  */
void randomDouble(double* ptr, int memsize)
{
	for (int i = 0;i <  memsize / sizeof(double); i++)
	{
		ptr[i] = (double)rand() / RAND_MAX * 1.0 - 1.0;
	}
}

/* ���� ���������� �� GPU */
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	
	float Cvalue = 0.0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > A.height || col > B.width) 
	{
		return;
	}
	for (int e = 0; e < A.width; ++e)
	{
		Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
	}

	C.elements[row * C.width + col] = Cvalue;
}




unsigned status;




int main()
{

	/* ���� ��� ����������, ������� ����� ������� */
	printf("CUDA devices:\n");
	int nDevices = 0;
	hipGetDeviceCount(&nDevices);
	if (!nDevices)
	{
		printf("Empty!\n");
		system("pause");
		return 0;
	}
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("\tDevice Number: %d\n", i);
		printf("\tDevice name: %s\n", prop.name);
		printf("\tPCI bus ID of the device: %d\n", prop.pciBusID);
		printf("\tPCI device ID of the device: %d", prop.pciDeviceID);
	}

	/* �������� ������ */

	while (1)
	{
		printf("\nEnter device number:"); scanf("%d", &nDevices);
		status = hipSetDevice(nDevices);

		if (status == hipSuccess)
		{
			break;
		}
		else
		{
			printf("Wrong number!\n");
		}
	}


	// ---


	int size;
	int mem_size;
	
	/* ������ �������, ������� ����� ��������� */
	while (1)
	{
		printf("Enter matrix size for mul: ");
		scanf("%d", &size);

		if (size <= 2)
		{
			printf("Wrong size!\n");
		}
		else
		{
			break;
		}
	}


	mem_size = size * size * sizeof(double);

	Matrix A;
	Matrix B;
	Matrix C;

	Matrix HOST;

	A.height = size; A.width = size;
	B.width = size; B.height = size;
	C.width = size; C.height = size;
	HOST.width = size; HOST.height = size;

	/* �������� ������ */
	status |= (unsigned)hipMalloc(&A.elements, mem_size);
	status |= (unsigned)hipMalloc(&B.elements, mem_size);
	status |= (unsigned)hipMalloc(&C.elements, mem_size);

	if (status)
	{
		printf("Cannot allocate memory on GPU!\n");
		system("pause");
		return 0;
	}
	else
	{
		printf("Memory on GPU succ. allocated.\n");
	}

	

	HOST.elements = (double*)malloc(mem_size);

	if (!HOST.elements)
	{
		printf("Cannot allocate memory on HOST!\n");
		system("pause");
		return 0;
	}
	else
	{
		printf("Memory on HOST succ. allocated.\n");
	}


	// ---

	printf("Data size:%d\n", mem_size);
	randomDouble(HOST.elements, mem_size);
	printf("Data generated succ.\n");

	// ----
	
	printf("Begin work...\n");
	while (1)
	{

		unsigned tm = clock();

		/* ������ ������ ����-���� */
		hipMemcpy(A.elements, HOST.elements, mem_size, hipMemcpyHostToDevice);
		hipMemcpy(A.elements, HOST.elements, mem_size, hipMemcpyHostToDevice);

		/* ������� �������, ��� �� ������� ��� */
		MatMulKernel << <size, size >> >(A, B, C);
		hipMemcpy(HOST.elements, C.elements, mem_size, hipMemcpyDeviceToHost);


		tm = clock() - tm;

		/* ������� ����� */
		printf("Runtime: %u ms\n", tm);


	}

	/* ������ ������� ����, ��� ����� �������� � ��� ����� */

	printf("\n");
	system("pause");
    return 0;
}